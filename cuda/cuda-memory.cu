#include "cuda-memory.h"

void cudaAlloc(void** p, size_t size) {
    cudaSafeCall(hipMalloc(p, size));
}

void cudaCopy(void* dest, void* src, size_t size, hipMemcpyKind type) {
    cudaSafeCall(hipMemcpy(dest, src, size, type));
}